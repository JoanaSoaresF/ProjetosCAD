/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#ifdef PNG
#include "pngwriter.h"
#endif

#define BLOCK_SIZE 16
#define NUM_ITERATIONS 10
#define VERSION "V1 - CUDA with memory transfer every step"

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i * width + j;
}

void __host__ __device__ initTemp(float *T, int h, int w)
{
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}
/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors)
{
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++)
    {
        for (int c = 0; c < width; c++)
        {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}

/* write heat map image
 */
void writeTemp(float *T, int h, int w, int n)
{
    char filename[64];
#ifdef PNG
    sprintf(filename, "../images/v1/heat_%06d.pgm", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "../images/v1/heat_%06d.pgm", n);
    FILE *f = fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}

__global__ void evolve_kernel(const float *Tn, float *Tnp1, const int nx, const int ny, const float a, const float h2, const float dt)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > 0 && i < nx - 1)
    {
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if (j > 0 && j < ny - 1)
        {
            const int index = getIndex(i, j, ny);
            float tij = Tn[index];
            float tim1j = Tn[getIndex(i - 1, j, ny)];
            float tijm1 = Tn[getIndex(i, j - 1, ny)];
            float tip1j = Tn[getIndex(i + 1, j, ny)];
            float tijp1 = Tn[getIndex(i, j + 1, ny)];

            // Explicit scheme
            Tnp1[index] = tij + a * dt * ((tim1j + tip1j + tijm1 + tijp1 - 4.0 * tij) / h2);
        }
    }
}

double timedif(struct timespec *t, struct timespec *t0)
{
    return (t->tv_sec - t0->tv_sec) + 1.0e-9 * (double)(t->tv_nsec - t0->tv_nsec);
}

int main()
{
    const int nx = 200;             // Width of the area
    const int ny = 200;             // Height of the area
    const float a = 0.5;            // Diffusion constant
    const float h = 0.005;          // h=dx=dy  grid spacing
    const int numSteps = 100000;    // Number of time steps to simulate (time=numSteps*dt)
    const int outputEvery = 100000; // How frequently to write output image

    const float h2 = h * h;

    const float dt = h2 / (4.0 * a); // Largest stable time step

    int numElements = nx * ny;
    // Allocate two sets of data for current and next timesteps
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(nx / threadsPerBlock.x + 1, ny / threadsPerBlock.y + 1);

    printf("--------------------------------------------------------------------------------------------\n");
    printf("VERSION: %s \n"
           "GENERAL PROBLEM:\n"
           "\tGrid: %d x %d\n"
           "\tGrid spacing(h): %f\n"
           "\tDiffusion constant: %f\n"
           "\tNumber of steps: %d\n"
           "\tOutput: %d steps\n"
           "CUDA PARAMETERS:\n"
           "\tThreads Per Block: %d x %d\n"
           "\tBlocks: %d x %d \n",
           VERSION, nx, ny, h, a, numSteps, outputEvery, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

    double totalTime = 0;
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        float *h_Tn = (float *)calloc(numElements, sizeof(float));
        float *h_Tnp1 = (float *)calloc(numElements, sizeof(float));

        // Initializing the data for T0
        initTemp(h_Tn, nx, ny);

        // Fill in the data on the next step to ensure that the boundaries are identical.
        memcpy(h_Tnp1, h_Tn, numElements * sizeof(float));

        float *d_Tn;
        float *d_Tnp1;
        hipMalloc((void **)&d_Tn, numElements * sizeof(float));
        hipMalloc((void **)&d_Tnp1, numElements * sizeof(float));

        writeTemp(h_Tn, nx, ny, 0);

        // Timing
        // clock_t start = clock();
        struct timespec start, finish;
        clock_gettime(CLOCK_MONOTONIC, &start);

        // Main loop

        for (int n = 0; n <= numSteps; n++)
        {
            hipMemcpy(d_Tn, h_Tn, numElements * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_Tnp1, h_Tnp1, numElements * sizeof(float), hipMemcpyHostToDevice);

            evolve_kernel<<<numBlocks, threadsPerBlock>>>(d_Tn, d_Tnp1, nx, ny, a, h2, dt);

            hipMemcpy(h_Tn, d_Tn, numElements * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(h_Tnp1, d_Tnp1, numElements * sizeof(float), hipMemcpyDeviceToHost);

            // Check if any error occurred during execution
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }

            // Write the output if needed
            if ((n + 1) % outputEvery == 0)
                writeTemp(h_Tnp1, nx, ny, n + 1);

            // Swapping the pointers for the next timestep
            float *t = h_Tn;
            h_Tn = h_Tnp1;
            h_Tnp1 = t;
        }

        // Timing
        // clock_t finish = clock();
        clock_gettime(CLOCK_MONOTONIC, &finish);
        double time = timedif(&finish, &start);
        totalTime += time;
        printf("Iteration %d took %f seconds\n", i, time);

        // Release the memory
        free(h_Tn);
        free(h_Tnp1);

        hipFree(d_Tn);
        hipFree(d_Tnp1);
    }

    printf("\nAverage time: %f\n\n", totalTime / (double)NUM_ITERATIONS);
    printf("--------------------------------------------------------------------------------------------\n");

    return 0;
}
