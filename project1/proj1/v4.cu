/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#ifdef PNG
#include "pngwriter.h"
#endif

#define NUM_ITERATIONS 10
#define BLOCK_SIZE 16
#define STREAMCOUNT_X 8
#define STREAMCOUNT_Y 8
//#define STREAMCOUNT = STREAMCOUNT_X * STREAMCOUNT_Y

/* Convert 2D index layout to Tnrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the Tnrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i * width + j;
}

void __host__ __device__ initTemp(float *T, int h, int w)
{
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}
/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors)
{
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++)
    {
        for (int c = 0; c < width; c++)
        {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}

/* write heat map image
 */
void writeTemp(float *T, int h, int w, int n)
{
    char filename[64];
#ifdef PNG
    sprintf(filename, "../images/v4/heat_%06d.pgm", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "../images/v4/heat_%06d.pgm", n);
    FILE *f = fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}

__global__ void evolve_kernel(int offsetX, int offsetY, const float *Tn, float *Tnp1, const int nx, const int ny, const float a, const float h2, const float dt)
{
    __shared__ float s_Tn[(BLOCK_SIZE + 2) * (BLOCK_SIZE + 2)];
    int i = offsetX + threadIdx.x + blockIdx.x * blockDim.x;
    int j = offsetY + threadIdx.y + blockIdx.y * blockDim.y;

    int s_i = threadIdx.x + 1;
    int s_j = threadIdx.y + 1;
    int s_ny = BLOCK_SIZE + 2;

    // Load data into shared memory
    // Central square
    s_Tn[getIndex(s_i, s_j, s_ny)] = Tn[getIndex(i, j, ny)];
    // Top border
    if (s_j == 1 && i != 0 && i < ny)
    {
        s_Tn[getIndex(s_i, 0, s_ny)] = Tn[getIndex(i, blockIdx.y * blockDim.y - 1, ny)];
    }
    // Bottom border
    if (s_j == BLOCK_SIZE && i != 0 && i < ny)
    {
        s_Tn[getIndex(s_i, BLOCK_SIZE + 1, s_ny)] = Tn[getIndex(i, (blockIdx.y + 1) * blockDim.y, ny)];
    }
    // Left border
    if (s_i == 1 && j != 0 && j < ny)
    {
        s_Tn[getIndex(0, s_j, s_ny)] = Tn[getIndex(blockIdx.x * blockDim.x - 1, j, ny)];
    }
    // Right border
    if (s_i == BLOCK_SIZE && j != 0 && j < ny)
    {
        s_Tn[getIndex(BLOCK_SIZE + 1, s_j, s_ny)] = Tn[getIndex((blockIdx.x + 1) * blockDim.x, j, ny)];
    }

    // Make sure all the data is loaded before computing
    __syncthreads();
    if (i > 0 && i < nx - 1)
    {
        if (j > 0 && j < ny - 1)
        {

            float tij = s_Tn[getIndex(s_i, s_j, s_ny)];
            float tim1j = s_Tn[getIndex(s_i - 1, s_j, s_ny)];
            float tijm1 = s_Tn[getIndex(s_i, s_j - 1, s_ny)];
            float tip1j = s_Tn[getIndex(s_i + 1, s_j, s_ny)];
            float tijp1 = s_Tn[getIndex(s_i, s_j + 1, s_ny)];

            // Explicit scheme
            const int index = getIndex(i, j, ny);
            Tnp1[index] = tij + a * dt * ((tim1j + tip1j + tijm1 + tijp1 - 4.0 * tij) / h2);
        }
    }
}

int main()
{
    const int nx = 200;             // Width of the area
    const int ny = 200;             // Height of the area
    const float a = 0.5;            // Diffusion constant
    const float h = 0.005;          // h=dx=dy  grid spacing
    const int numSteps = 100000;    // Number of time steps to simulate (time=numSteps*dt)
    const int outputEvery = 100000; // How frequently to write output image

    const float h2 = h * h;

    const float dt = h2 / (4.0 * a); // Largest stable time step


    int numElements = nx * ny;
    
    //Streams
    //STREAMCOUNT;
    int streamSize = ceil((nx / STREAMCOUNT_X)) * ceil((ny/ STREAMCOUNT_X)) ;
    int streamSizeX = ceil((nx) / STREAMCOUNT_X);
    int streamSizeY = ceil((ny) / STREAMCOUNT_Y); 
    
    
    // Allocate two sets of data for current and next timesteps

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(nx / threadsPerBlock.x + 1, ny / threadsPerBlock.y + 1);

    double totalTime = 0;
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        float *h_Tn = (float *)calloc(numElements, sizeof(float));
        float *h_Tnp1 = (float *)calloc(numElements, sizeof(float));

        // Initializing the data for T0
        initTemp(h_Tn, nx, ny);

        // Fill in the data on the next step to ensure that the boTndaries are identical.
        memcpy(h_Tnp1, h_Tn, numElements * sizeof(float));

        float *d_Tn;
        float *d_Tnp1;
        hipMalloc((void **)&d_Tn, numElements * sizeof(float));
        hipMalloc((void **)&d_Tnp1, numElements * sizeof(float));


        writeTemp(h_Tn, nx, ny, 0);

        //Stream cudaMemcpyAsync + evolve_kernel (1st round)
        int offset = 0;
        int offsetX = 0;
        int offsetY = 0;
        hipStream_t* streams = (hipStream_t*)malloc(STREAMCOUNT_X * STREAMCOUNT_Y * sizeof(hipStream_t));

        for (int s = 0; s < STREAMCOUNT_X * STREAMCOUNT_Y; s++)
        {
            hipStreamCreate(&streams[s]);
        }


        // Timing
        clock_t start = clock();


        for(int ystream = 0; ystream < STREAMCOUNT_Y; ystream++){
            offsetY = ystream * streamSizeY;

            for(int xstream = 0; xstream < STREAMCOUNT_X; xstream++)
            {
                offsetX = xstream * streamSizeX;

                int streamNr = ystream * STREAMCOUNT_X + xstream;

                for (int cy = 0; cy<streamSizeY+2; cy++){
                    offset = offsetY * nx + offsetX;

                    //printf("creating stream: ", streamNr);
                    //cudaStreamCreate(&streams[streamNr]);

                    //printf("Copying to gpu streamX: %d \n" + xstream);
                    hipMemcpyAsync(&d_Tn[offset], &h_Tn[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, streams[streamNr]);
                    hipMemcpyAsync(&d_Tnp1[offset], &h_Tnp1[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, streams[streamNr]);

                }
                evolve_kernel<<<streamSize/BLOCK_SIZE, threadsPerBlock, 0, streams[i]>>>(offsetX, offsetY, d_Tn, d_Tnp1, nx, ny, a, h2, dt);

            }

        }
        //cudaMemcpy(d_Tn, h_Tn, numElements * sizeof(float), cudaMemcpyHostToDevice);
        //cudaMemcpy(d_Tnp1, h_Tnp1, numElements * sizeof(float), cudaMemcpyHostToDevice);
        
        // Main loop
        for (int n = 1; n <= numSteps; n++) //Streams 0 ->> 1
        {

            evolve_kernel<<<numBlocks, threadsPerBlock>>>(0, 0, d_Tn, d_Tnp1, nx, ny, a, h2, dt);

            // Check if any error occurred during execution
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }

            // Write the output if needed
            if ((n + 1) % outputEvery == 0)
            {
                hipMemcpy(h_Tn, d_Tn, numElements * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(h_Tnp1, d_Tnp1, numElements * sizeof(float), hipMemcpyDeviceToHost);
                if (errorCode != hipSuccess)
                {
                    printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                    exit(0);
                }
                writeTemp(h_Tnp1, nx, ny, n + 1);
            }

            // Swapping the pointers for the next timestep
            float *t = d_Tn;
            d_Tn = d_Tnp1;
            d_Tnp1 = t;
        }

        // Timing
        clock_t finish = clock();
        double time = (double)(finish - start) / CLOCKS_PER_SEC;
        totalTime += time;
        printf("Iteration %d took %f seconds\n", i, time);

        // Release the memory
        free(h_Tn);
        free(h_Tnp1);

        hipFree(d_Tn);
        hipFree(d_Tnp1);

        for (int s = 0; s < STREAMCOUNT_X * STREAMCOUNT_Y; ++s)
        {
            hipStreamDestroy(streams[s]);
        }

    }

    printf("Average time: %f\n", totalTime / (double)NUM_ITERATIONS);

    return 0;
}


