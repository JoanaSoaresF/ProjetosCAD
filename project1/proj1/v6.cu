/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#ifdef PNG
#include "pngwriter.h"
#endif

#define NUM_ITERATIONS 10
#define BLOCK_SIZE 16

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i * width + j;
}

void __host__ __device__ initTemp(float *T, int h, int w)
{
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}
/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors)
{
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++)
    {
        for (int c = 0; c < width; c++)
        {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}

/* write heat map image
 */
void writeTemp(float *T, int h, int w, int n)
{
    char filename[64];
#ifdef PNG
    sprintf(filename, "../images/v6/heat_%06d.pgm", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "../images/v6/heat_%06d.pgm", n);
    FILE *f = fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}

__global__ void evolve_kernel(const float *Tn, float *Tnp1, const int nx, const int ny, const float a, const float h2, const float dt)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i > 0 && i < nx - 1)
    {
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if (j > 0 && j < ny - 1)
        {
            const int index = getIndex(i, j, ny);
            float tij = Tn[index];
            float tim1j = Tn[getIndex(i - 1, j, ny)];
            float tijm1 = Tn[getIndex(i, j - 1, ny)];
            float tip1j = Tn[getIndex(i + 1, j, ny)];
            float tijp1 = Tn[getIndex(i, j + 1, ny)];

            // Explicit scheme
            Tnp1[index] = tij + a * dt * ((tim1j + tip1j + tijm1 + tijp1 - 4.0 * tij) / h2);
        }
    }
}

int main()
{
    const int nx = 200;           // Width of the area
    const int ny = 200;           // Height of the area
    const float a = 0.5;          // Diffusion constant
    const float h = 0.005;        // h=dx=dy  grid spacing
    const int numSteps = 100000;  // Number of time steps to simulate (time=numSteps*dt)
    const int outputEvery = 1000; // How frequently to write output image

    const float h2 = h * h;

    const float dt = h2 / (4.0 * a); // Largest stable time step

    int numElements = nx * ny;

    // Allocate two sets of data for current and next timesteps

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(nx / threadsPerBlock.x + 1, ny / threadsPerBlock.y + 1);

    double totalTime = 0;
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        float *h_Tn = (float *)calloc(numElements, sizeof(float));
        float *h_Tnp1 = (float *)calloc(numElements, sizeof(float));

        // Initializing the data for T0
        initTemp(h_Tn, nx, ny);

        // Fill in the data on the next step to ensure that the boundaries are identical.
        memcpy(h_Tnp1, h_Tn, numElements * sizeof(float));

        float *d_Tn;
        float *d_Tnp1;
        hipMalloc((void **)&d_Tn, numElements * sizeof(float));
        hipMalloc((void **)&d_Tnp1, numElements * sizeof(float));
        hipMemcpy(d_Tn, h_Tn, numElements * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_Tnp1, h_Tnp1, numElements * sizeof(float), hipMemcpyHostToDevice);

        writeTemp(h_Tn, nx, ny, 0);

        //    Create streams
        hipStream_t stream;
        hipStreamCreate(&stream);

        hipStream_t streamKernel;
        hipStreamCreate(&streamKernel);

        hipEvent_t event;
        hipEventCreate(&event);

        hipEvent_t eventKernel;
        hipEventCreate(&eventKernel);

        // Timing
        clock_t start = clock();

        // Main loop

        for (int n = 0; n <= numSteps; n++)
        {

            
            evolve_kernel<<<numBlocks, threadsPerBlock>>>(d_Tn, d_Tnp1, nx, ny, a, h2, dt);
            // cudaDeviceSynchronize();

            // Check if any error occurred during execution
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d in iteration %d: %s\n", errorCode, n, hipGetErrorString(errorCode));
                exit(0);
            }

            // Write the output if needed
            if ((n + 1) % outputEvery == 0 && n != 0)
            {
                hipError_t error = hipMemcpyAsync(h_Tn, d_Tn, numElements * sizeof(float), 
                hipMemcpyDeviceToHost, stream);

                if (error != hipSuccess)
                {
                    printf("Cuda error %d in iteration dentro do if depois memo %d: %s\n", error, n, hipGetErrorString(error));
                    exit(0);
                }
                hipEventRecord(event, stream);
                errorCode = hipGetLastError();
                if (errorCode != hipSuccess)
                {
                    printf("Cuda error %d in iteration dentro do ifdepois do event %d: %s\n", errorCode, n, hipGetErrorString(errorCode));
                    exit(0);
                }

                // TODO Temos que garantir que temos h_Tn antes de escrever
                evolve_kernel<<<numBlocks, threadsPerBlock, 0, streamKernel>>>(d_Tn, d_Tnp1, nx, ny, a, h2, dt);
                hipEventRecord(eventKernel, streamKernel);

                n++;

                hipStreamWaitEvent(stream, event);
                writeTemp(h_Tn, nx, ny, n + 1);
                hipStreamWaitEvent(streamKernel, eventKernel);
            }

            // Swapping the pointers for the next timestep
            float *t = d_Tn;
            d_Tn = d_Tnp1;
            d_Tnp1 = t;
        }

        // Timing
        clock_t finish = clock();
        double time = (double)(finish - start) / CLOCKS_PER_SEC;
        totalTime += time;
        printf("Iteration %d took %f seconds\n", i, time);

        // Release the memory
        free(h_Tn);
        free(h_Tnp1);

        hipFree(d_Tn);
        hipFree(d_Tnp1);

        hipStreamDestroy(stream);
    }

    printf("Average time: %f\n", totalTime / (double)NUM_ITERATIONS);

    return 0;
}
