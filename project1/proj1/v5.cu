/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#ifdef PNG
#include "pngwriter.h"
#endif
// TODO run

#define NUM_ITERATIONS 10
#define BLOCK_SIZE 16
#define STREAMCOUNT_X 4
#define STREAMCOUNT_Y 4
#define VERSION "V5 - streams without shared memory - initial transfer"
//#define STREAMCOUNT = STREAMCOUNT_X * STREAMCOUNT_Y

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i * width + j;
}

void __host__ __device__ initTemp(float *T, int h, int w)
{
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}
/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors)
{
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++)
    {
        for (int c = 0; c < width; c++)
        {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}

/* write heat map image
 */
void writeTemp(float *T, int h, int w, int n)
{
    char filename[64];
#ifdef PNG
    sprintf(filename, "../images/v5/heat_%06d.pgm", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "../images/v5/heat_%06d.pgm", n);
    FILE *f = fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}

__global__ void evolve_kernel(int offsetX, int offsetY, const float *Tn, float *Tnp1, const int nx, const int ny, const float a, const float h2, const float dt)
{
    int i = offsetX + threadIdx.x + blockIdx.x * blockDim.x;
    if (i > 0 && i < nx - 1)
    {
        int j = offsetY + threadIdx.y + blockIdx.y * blockDim.y;
        if (j > 0 && j < ny - 1)
        {
            const int index = getIndex(i, j, ny);
            float tij = Tn[index];
            float tim1j = Tn[getIndex(i - 1, j, ny)];
            float tijm1 = Tn[getIndex(i, j - 1, ny)];
            float tip1j = Tn[getIndex(i + 1, j, ny)];
            float tijp1 = Tn[getIndex(i, j + 1, ny)];

            // Explicit scheme
            Tnp1[index] = tij + a * dt * ((tim1j + tip1j + tijm1 + tijp1 - 4.0 * tij) / h2);
        }
    }
}

double timedif(struct timespec *t, struct timespec *t0)
{
    return (t->tv_sec - t0->tv_sec) + 1.0e-9 * (double)(t->tv_nsec - t0->tv_nsec);
}

int main()
{
    const int nx = 200;             // Width of the area
    const int ny = 200;             // Height of the area
    const float a = 0.5;            // Diffusion constant
    const float h = 0.005;          // h=dx=dy  grid spacing
    const int numSteps = 100000;    // Number of time steps to simulate (time=numSteps*dt) .  1000000
    const int outputEvery = 100000; // How frequently to write output image

    const float h2 = h * h;

    const float dt = h2 / (4.0 * a); // Largest stable time step

    int numElements = nx * ny;

    // Streams
    // STREAMCOUNT;
    int streamSize = ceil((nx / STREAMCOUNT_X)) * ceil((ny / STREAMCOUNT_X));
    int streamSizeX = ceil((nx) / STREAMCOUNT_X);
    int streamSizeY = ceil((ny) / STREAMCOUNT_Y);

    // Allocate two sets of data for current and next timesteps

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(nx / threadsPerBlock.x + 1, ny / threadsPerBlock.y + 1);
    int numBlocksStream = streamSize / BLOCK_SIZE;

    printf("--------------------------------------------------------------------------------------------\n");
    printf("VERSION: %s \n"
           "GENERAL PROBLEM:\n"
           "\tGrid: %d x %d\n"
           "\tGrid spacing(h): %f\n"
           "\tDiffusion constant: %f\n"
           "\tNumber of steps: %d\n "
           "\tOutput: %d steps\n"
           "CUDA PARAMETERS:\n"
           "\tThreads Per Block: %d x %d\n"
           "\tBlocks: %d x %d\n"
           "STREAMS:\n"
           "\tNumber of streams: %d x %d\n"
           "\tStream Size: %d\n"
           "\tBlock size with stream: %d\n\n",
           VERSION, nx, ny, h, a, numSteps, outputEvery, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y, STREAMCOUNT_X, STREAMCOUNT_Y, streamSize, numBlocksStream);

    double totalTime = 0;
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        float *h_Tn = (float *)calloc(numElements, sizeof(float));
        float *h_Tnp1 = (float *)calloc(numElements, sizeof(float));

        // Initializing the data for T0
        initTemp(h_Tn, nx, ny);

        // Fill in the data on the next step to ensure that the boundaries are identical.
        memcpy(h_Tnp1, h_Tn, numElements * sizeof(float));

        float *d_Tn;
        float *d_Tnp1;
        hipMalloc((void **)&d_Tn, numElements * sizeof(float));
        hipMalloc((void **)&d_Tnp1, numElements * sizeof(float));

        writeTemp(h_Tn, nx, ny, 0);

        // Stream cudaMemcpyAsync + evolve_kernel (1st round)
        int offset = 0;
        int offsetX = 0;
        int offsetY = 0;
        hipStream_t *streams = (hipStream_t *)malloc(STREAMCOUNT_X * STREAMCOUNT_Y * sizeof(hipStream_t));

        for (int s = 0; s < STREAMCOUNT_X * STREAMCOUNT_Y; s++)
        {
            hipStreamCreate(&streams[s]);
        }

        // Timing
        // clock_t start = clock();
        struct timespec start, finish;
        clock_gettime(CLOCK_MONOTONIC, &start);

        for (int ystream = 0; ystream < STREAMCOUNT_Y; ystream++)
        {
            offsetY = ystream * streamSizeY;

            for (int xstream = 0; xstream < STREAMCOUNT_X; xstream++)
            {
                offsetX = xstream * streamSizeX;

                int streamNr = ystream * STREAMCOUNT_X + xstream;

                for (int cy = 0; cy < streamSizeY + 2; cy++)
                {
                    offset = offsetY * nx + offsetX;

                    hipMemcpyAsync(&d_Tn[offset], &h_Tn[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, streams[streamNr]);
                    hipMemcpyAsync(&d_Tnp1[offset], &h_Tnp1[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, streams[streamNr]);
                }
                evolve_kernel<<<numBlocksStream, threadsPerBlock, 0, streams[streamNr]>>>(offsetX, offsetY, d_Tn, d_Tnp1, nx, ny, a, h2, dt);
            }
        }

        // Main loop

        for (int n = 1; n <= numSteps; n++)
        {

            evolve_kernel<<<numBlocks, threadsPerBlock>>>(0, 0, d_Tn, d_Tnp1, nx, ny, a, h2, dt);

            // Check if any error occurred during execution
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }

            // Write the output if needed
            if ((n + 1) % outputEvery == 0)
            {
                hipMemcpy(h_Tn, d_Tn, numElements * sizeof(float), hipMemcpyDeviceToHost);
                hipMemcpy(h_Tnp1, d_Tnp1, numElements * sizeof(float), hipMemcpyDeviceToHost);
                writeTemp(h_Tnp1, nx, ny, n + 1);
            }

            // Swapping the pointers for the next timestep
            float *t = d_Tn;
            d_Tn = d_Tnp1;
            d_Tnp1 = t;
        }

        // Timing
        clock_gettime(CLOCK_MONOTONIC, &finish);
        double time = timedif(&finish, &start);
        totalTime += time;
        printf("Iteration %d took %f seconds\n", i, time);

        // Release the memory
        free(h_Tn);
        free(h_Tnp1);

        hipFree(d_Tn);
        hipFree(d_Tnp1);

        for (int s = 0; s < STREAMCOUNT_X * STREAMCOUNT_Y; ++s)
        {
            hipStreamDestroy(streams[s]);
        }
    }

    printf("\nAverage time: %f\n\n", totalTime / (double)NUM_ITERATIONS);
    printf("--------------------------------------------------------------------------------------------\n");

    return 0;
}
