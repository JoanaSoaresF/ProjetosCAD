/*
 * Based on CSC materials from:
 *
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#ifdef PNG
#include "pngwriter.h"
#endif

#define BLOCK_SIZE 16
#define NUM_ITERATIONS 1
#define STREAMCOUNT_X 4
#define STREAMCOUNT_Y 4
#define VERSION "V8 - Streams with comunnication every step - same loop"

/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 *
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i * width + j;
}

void __host__ __device__ initTemp(float *T, int h, int w)
{
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}
/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors)
{
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++)
    {
        for (int c = 0; c < width; c++)
        {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}

/* write heat map image
 */
void writeTemp(float *T, int h, int w, int n)
{
    char filename[64];
#ifdef PNG
    sprintf(filename, "../images/v8/heat_%06d.pgm", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "../images/v8/heat_%06d.pgm", n);
    FILE *f = fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}
__global__ void evolve_kernel(int offsetX, int offsetY, const float *Tn, float *Tnp1, const int nx, const int ny, const float a, const float h2, const float dt)
{
    int i = offsetX + threadIdx.x + blockIdx.x * blockDim.x;
    if (i > 0 && i < nx - 1)
    {
        int j = offsetY + threadIdx.y + blockIdx.y * blockDim.y;
        if (j > 0 && j < ny - 1)
        {
            const int index = getIndex(i, j, ny);
            float tij = Tn[index];
            float tim1j = Tn[getIndex(i - 1, j, ny)];
            float tijm1 = Tn[getIndex(i, j - 1, ny)];
            float tip1j = Tn[getIndex(i + 1, j, ny)];
            float tijp1 = Tn[getIndex(i, j + 1, ny)];

            // Explicit scheme
            Tnp1[index] = tij + a * dt * ((tim1j + tip1j + tijm1 + tijp1 - 4.0 * tij) / h2);
        }
    }
}

double timedif(struct timespec *t, struct timespec *t0)
{
    return (t->tv_sec - t0->tv_sec) + 1.0e-9 * (double)(t->tv_nsec - t0->tv_nsec);
}

int main()
{
    const int nx = 200;             // Width of the area
    const int ny = 200;             // Height of the area
    const float a = 0.5;            // Diffusion constant
    const float h = 0.005;          // h=dx=dy  grid spacing
    const int numSteps = 100000;    // Number of time steps to simulate (time=numSteps*dt)
    const int outputEvery = 100000; // How frequently to write output image

    const float h2 = h * h;

    const float dt = h2 / (4.0 * a); // Largest stable time step

    int numElements = nx * ny;
    // Allocate two sets of data for current and next timesteps
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(nx / threadsPerBlock.x + 1, ny / threadsPerBlock.y + 1);

    double totalTime = 0;
    printf("--------------------------------------------------------------------------------------------\n");
    printf("VERSION: %s \n"
           "GENERAL PROBLEM:\n"
           "\tGrid: %d x %d\n"
           "\tGrid spacing(h): %f\n"
           "\tDiffusion constant: %f\n"
           "\tNumber of steps: %d\n "
           "\tOutput: %d steps\n"
           "CUDA PARAMETERS:\n"
           "\tThreads Per Block: %d x %d\n"
           "\tBlocks: %d x %d \n\n"
           "STREAMS:\n"
           "\tNumber of streams: %d x %d\n"
           "\tStream Size: %d\n\n",
           VERSION, nx, ny, h, a, numSteps, outputEvery, threadsPerBlock.x, threadsPerBlock.y, numBlocks, STREAMCOUNT_X, STREAMCOUNT_Y, 0);

    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        float *h_Tn = (float *)calloc(numElements, sizeof(float));
        float *h_Tnp1 = (float *)calloc(numElements, sizeof(float));

        // Initializing the data for T0
        initTemp(h_Tn, nx, ny);

        // Fill in the data on the next step to ensure that the boundaries are identical.
        memcpy(h_Tnp1, h_Tn, numElements * sizeof(float));

        float *d_Tn;
        float *d_Tnp1;
        hipMalloc((void **)&d_Tn, numElements * sizeof(float));
        hipMalloc((void **)&d_Tnp1, numElements * sizeof(float));

        writeTemp(h_Tn, nx, ny, 0);

        // Streams
        // STREAMCOUNT;
        int streamSize = ceil((nx / STREAMCOUNT_X)) * ceil((ny / STREAMCOUNT_X));
        int streamSizeX = ceil((nx) / STREAMCOUNT_X);
        int streamSizeY = ceil((ny) / STREAMCOUNT_Y);

        //    Create streams
        int nStreams = STREAMCOUNT_X * STREAMCOUNT_Y;
        hipStream_t *stream = (hipStream_t *)malloc(nStreams * sizeof(hipStream_t));
        // cudaStream_t streamRecive[nStreams];

        for (int s = 0; s < nStreams; s++)
        {
            hipStreamCreate(&stream[s]);
        }

        // Timing
        // clock_t start = clock();
        struct timespec start, finish;
        clock_gettime(CLOCK_MONOTONIC, &start);

        // Main loop
        int offsetX, offsetY, offset;

        for (int n = 0; n <= numSteps; n++)
        {
            // Copy Tn to device
            for (int ystream = 0; ystream < STREAMCOUNT_Y; ystream++)
            {
                offsetY = ystream * streamSizeY;

                for (int xstream = 0; xstream < STREAMCOUNT_X; xstream++)
                {
                    offsetX = xstream * streamSizeX;

                    int streamNr = ystream * STREAMCOUNT_X + xstream;

                    for (int cy = 0; cy < streamSizeY + 2; cy++)
                    {
                        offset = offsetY * nx + offsetX;

                        // cudaStreamCreate(&streams[streamNr]);

                        // printf("Copying to gpu streamX: %d \n" + xstream);
                        hipMemcpyAsync(&d_Tn[offset], &h_Tn[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, stream[streamNr]);
                        hipMemcpyAsync(&d_Tnp1[offset], &h_Tnp1[offset], (streamSizeX) * sizeof(float), hipMemcpyHostToDevice, stream[streamNr]);
                    }
                    evolve_kernel<<<streamSize / BLOCK_SIZE, threadsPerBlock, 0, stream[streamNr]>>>(offsetX, offsetY, d_Tn, d_Tnp1, nx, ny, a, h2, dt);
                }
            }
            // cudaDeviceSynchronize();

            hipMemcpy(h_Tn, d_Tn, numElements * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(h_Tnp1, d_Tnp1, numElements * sizeof(float), hipMemcpyDeviceToHost);

            // Check if any error occurred during execution
            hipError_t errorCode = hipGetLastError();
            if (errorCode != hipSuccess)
            {
                printf("Cuda error %d: %s\n", errorCode, hipGetErrorString(errorCode));
                exit(0);
            }

            // Write the output if needed
            if ((n + 1) % outputEvery == 0)
                writeTemp(h_Tnp1, nx, ny, n + 1);

            // Swapping the pointers for the next timestep
            float *t = h_Tn;
            h_Tn = h_Tnp1;
            h_Tnp1 = t;
        }

        // Timing
        // clock_t finish = clock();
        // double time = (double)(finish - start) / CLOCKS_PER_SEC;
        clock_gettime(CLOCK_MONOTONIC, &finish);
        double time = timedif(&finish, &start);
        totalTime += time;
        printf("Iteration %d took %f seconds\n", i, time);

        // Release the memory
        free(h_Tn);
        free(h_Tnp1);

        hipFree(d_Tn);
        hipFree(d_Tnp1);

        for (int i = 0; i < nStreams; ++i)
        {
            hipStreamDestroy(stream[i]);
        }
    }

    printf("\nAverage time: %f\n\n", totalTime / (double)NUM_ITERATIONS);
    printf("--------------------------------------------------------------------------------------------\n");

    return 0;
}
